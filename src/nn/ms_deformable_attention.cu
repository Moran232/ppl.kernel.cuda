#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/nn/ms_deformable_attention.h"
#include "ppl/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include "cudakernel/common/common.cuh"
#include "cudakernel/common/common.h"
#include <hip/hip_fp16.h>


#define CUDA_KERNEL_LOOP(i, n)                          \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;   \
      i < (n);                                          \
      i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N, const int num_threads)
{
  return (N + num_threads - 1) / num_threads;
}

template <typename scalar_t>
__device__ scalar_t ms_deform_attn_im2col_bilinear(const scalar_t* &bottom_data, 
                                                   const int &height, const int &width, const int &nheads, const int &channels,
                                                   const scalar_t &h, const scalar_t &w, const int &m, const int &c)
{
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
  {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
  {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
  {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
  {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__global__ void ms_deformable_im2col_gpu_kernel(const int n,
                                                const scalar_t *data_value, 
                                                const int64_t *data_spatial_shapes,
                                                const int64_t *data_level_start_index, 
                                                const scalar_t *data_sampling_loc,
                                                const scalar_t *data_attn_weight,
                                                const int batch_size, 
                                                const int spatial_size, 
                                                const int num_heads,
                                                const int channels, 
                                                const int num_levels,
                                                const int num_query,
                                                const int num_point,
                                                scalar_t *data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp; 
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    const int q_col = _temp % num_query;
    _temp /= num_query;
    const int b_col = _temp;

    scalar_t *data_col_ptr = data_col + index;
    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;
    scalar_t col = 0;
    
    for (int l_col=0; l_col < num_levels; ++l_col)
    {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const scalar_t *data_value_ptr = data_value + (data_value_ptr_init_offset + level_start_id * qid_stride);
      for (int p_col=0; p_col < num_point; ++p_col)
      {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w)
        {
          col += ms_deform_attn_im2col_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im, w_im, m_col, c_col) * weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
    }
    *data_col_ptr = col;
  }
}


ppl::common::RetCode PPLCUDAMSDeformAttnForwardImp(
    const hipStream_t &stream,
    const ppl::common::TensorShape *input_shape,
    const ppl::common::TensorShape *output_shape,
    const void *data, // float
    const void *spatial_shapes, // int64_t
    const void *level_start_index, // int64_t
    const void *sampling_loc, // float
    const void *attn_weight,  // float
    void* output,                  // float
    const int batch,
    int64_t im2col_step_, 
    const int spatial_size, 
    const int num_heads, 
    const int channels, 
    const int num_levels, 
    const int num_query, 
    const int num_point){


    auto per_value_size = spatial_size * num_heads * channels;
    auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
    auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;

    const int batch_n = im2col_step_;


    // output {batch/im2col_step_, batch_n, num_query, num_heads, channels}
    for(int n=0; n < batch/im2col_step_; ++n){

        auto data_value = (float*)data + n * im2col_step_ * per_value_size;
        auto data_spatial_shapes = (int64_t*)spatial_shapes;
        auto data_level_start_index = (int64_t*)level_start_index;
        auto data_sampling_loc = (float*)sampling_loc + n * im2col_step_ * per_sample_loc_size;
        auto data_attn_weight = (float*)attn_weight + n * im2col_step_ * per_attn_weight_size;
        auto data_col = (float*)output + n*(batch_n*num_query*num_heads*channels);

        auto batch_size = batch_n;

        int num_kernels = batch_size * num_query * num_heads * channels;
        int num_actual_kernels = batch_size * num_query * num_heads * channels;
        int num_threads = CUDA_NUM_THREADS;

        ms_deformable_im2col_gpu_kernel<float><<<
            GET_BLOCKS(num_actual_kernels, num_threads), num_threads,0, stream>>>(
        num_kernels, data_value, data_spatial_shapes, data_level_start_index, data_sampling_loc, data_attn_weight, 
        batch_size, spatial_size, num_heads, channels, num_levels, num_query, num_point, data_col);


    }


    return ppl::common::RC_SUCCESS;
}
